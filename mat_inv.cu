
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include<math.h>


#define cudacall(call)                                                                                                          \
    do                                                                                                                          \
    {                                                                                                                           \
        hipError_t err = (call);                                                                                               \
        if(hipSuccess != err)                                                                                                  \
        {                                                                                                                       \
            fprintf(stderr,"CUDA Error:\nFile = %s\nLine = %d\nReason = %s\n", __FILE__, __LINE__, hipGetErrorString(err));    \
            hipDeviceReset();                                                                                                  \
            exit(EXIT_FAILURE);                                                                                                 \
        }                                                                                                                       \
    }                                                                                                                           \
    while (0)

#define cublascall(call)                                                                                        \
    do                                                                                                          \
    {                                                                                                           \
        hipblasStatus_t status = (call);                                                                         \
        if(HIPBLAS_STATUS_SUCCESS != status)                                                                     \
        {                                                                                                       \
            fprintf(stderr,"CUBLAS Error:\nFile = %s\nLine = %d\nCode = %d\n", __FILE__, __LINE__, status);     \
            hipDeviceReset();                                                                                  \
            exit(EXIT_FAILURE);                                                                                 \
        }                                                                                                       \
                                                                                                                \
    }                                                                                                           \
    while(0)

////////////////////////////////////////////////////////////////////////
//
//
////////////////////////////////////////////////////////////////////////
void invert(float** src, float** dst, int n, int batchSize)
{
    	hipblasHandle_t handle;
    	cublascall(hipblasCreate(&handle));

    	int *P, *INFO;

    	cudacall(hipMalloc(&P, n * batchSize * sizeof(int)));
    	cudacall(hipMalloc(&INFO,  batchSize * sizeof(int)));

    	int lda = n;

    	float **A = (float **)malloc(batchSize*sizeof(float *));
    	float **A_d, *A_dflat;

    	cudacall(hipMalloc(&A_d,batchSize*sizeof(float *)));
    	cudacall(hipMalloc(&A_dflat, n*n*batchSize*sizeof(float)));

	A[0] = A_dflat;
    	for (int i = 1; i < batchSize; i++)
      		A[i] = A[i-1]+(n*n);

    	cudacall(hipMemcpy(A_d,A,batchSize*sizeof(float *),hipMemcpyHostToDevice));
   
 	for (int i = 0; i < batchSize; i++)
      		cudacall(hipMemcpy(A_dflat+(i*n*n), src[i], n*n*sizeof(float), hipMemcpyHostToDevice));


    	cublascall(hipblasSgetrfBatched(handle,n,A_d,lda,P,INFO,batchSize));


    	int INFOh[batchSize];
    	cudacall(hipMemcpy(INFOh,INFO,batchSize*sizeof(int),hipMemcpyDeviceToHost));

    	for (int i = 0; i < batchSize; i++)
      		if(INFOh[i]  != 0)
      		{
        		fprintf(stderr, "Factorization of matrix %d Failed: Matrix may be singular\n", i);
        		hipDeviceReset();
        		exit(EXIT_FAILURE);
      		}

    	float **C = (float **)malloc(batchSize*sizeof(float *));
    	float **C_d, *C_dflat;

    	cudacall(hipMalloc(&C_d,batchSize*sizeof(float *)));
    	cudacall(hipMalloc(&C_dflat, n*n*batchSize*sizeof(float)));
    	C[0] = C_dflat;
    	for (int i = 1; i < batchSize; i++)
      		C[i] = C[i-1] + (n*n);
    	cudacall(hipMemcpy(C_d,C,batchSize*sizeof(float *),hipMemcpyHostToDevice));
    	cublascall(hipblasSgetriBatched(handle,n,(const float **)A_d,lda,P,C_d,lda,INFO,batchSize));

    	cudacall(hipMemcpy(INFOh,INFO,batchSize*sizeof(int),hipMemcpyDeviceToHost));

    	for (int i = 0; i < batchSize; i++)
	      	if(INFOh[i] != 0)
      		{
        		fprintf(stderr, "Inversion of matrix %d Failed: Matrix may be singular\n", i);
        		hipDeviceReset();
      			exit(EXIT_FAILURE);
      		}
    	for (int i = 0; i < batchSize; i++)
      	cudacall(hipMemcpy(dst[i], C_dflat + (i*n*n), n*n*sizeof(float), hipMemcpyDeviceToHost));
    	
	hipFree(A_d); hipFree(A_dflat); free(A);
	hipFree(C_d); hipFree(C_dflat); free(C);
    	hipFree(P); hipFree(INFO); hipblasDestroy(handle);
}


////////////////////////////////////////////////////////////////////////
//
//
////////////////////////////////////////////////////////////////////////


__global__ void MatrixMulKernel(float *Md, float *Nd, float *Pd, int Width) 
{
    //2D Thread ID
    int col = blockIdx.x*blockDim.x+threadIdx.x;
    int row = blockIdx.y*blockDim.y+threadIdx.y;

    //Pvalue stores the Pd element that is computed by the thread
    float Pvalue = 0;
	if(col<Width && row < Width)
	{
		    for(int k = 0; k < Width ; ++k) 
		    {
		        float Mdelement = Md[row*Width + k];
		        float Ndelement = Nd[k*Width + col];
		        Pvalue += (Mdelement*Ndelement);
		
		    }
		    Pd[row*Width + col] = Pvalue;
	}
}



void mul(float* M,float* N,int Width)
{
		
	float * P = (float *) malloc(Width*Width*sizeof(float));
	float *Md, *Nd, *Pd;



	unsigned long int size = Width*Width*sizeof(float);
  

    //Transfer M and N to device memory
    	hipMalloc((void**)&Md, size);
    	hipMemcpy(Md,M,size,hipMemcpyHostToDevice);

    	hipMalloc((void**)&Nd, size);
    	hipMemcpy(Nd,N,size,hipMemcpyHostToDevice);

    	//Allocate P on the device
    	hipMalloc((void**)&Pd,size);

    	//Setup the execution configuration
    	dim3 dimBlock(Width,Width);
    	dim3 dimGrid(1,1);


	if (Width*Width > 1024)
	{
		//printf("\n\n enter inside if condi\n\n");
		
		dimGrid.x = (Width-1)/32+1;
        	dimGrid.y = (Width-1)/32+1;
	
		dimBlock.x = 32;
	        dimBlock.y = 32;



	}

     
    //Launch the device computation threads!
	MatrixMulKernel<<<dimGrid,dimBlock>>>(Md,Nd,Pd,Width);

    //Transfer P from device to host
    	hipMemcpy(P,Pd,size,hipMemcpyDeviceToHost);

    //Free device matrices
    	hipFree(Md);
    	hipFree(Nd);
    	hipFree(Pd);

	int i;

	fprintf(stdout,"\n\n");

	if(Width<11)
	{


		fprintf(stdout,"\n\nMatrix Multiplication, M x Inv(M) :\n\n");
		for(i = 0; i < Width*Width; i++)
   		{
			if(P[i])
				fprintf(stdout,"%10f ",P[i]) ;
			else
				fprintf(stdout,"%9f ",P[i]) ;
	        	



			if((i+1)%Width==0)
				fprintf(stdout,"\n");
		}
  

	}
	else
	{
		FILE *fp;	
	
		fp = fopen("Mat_Inv_out", "a");

 		if (!fp) 
		{
	    		fprintf(stderr, "Failed to open matAdata.\n");
	    		exit(1);
	  	}
		fprintf(fp,"\n\nMatrix Multiplication, M x Inv(M) :\n\n");
	 	for(i = 0; i < Width*Width; i++)
   		{	if(P[i])
				fprintf(fp,"%10f ",P[i]) ;
			else
				fprintf(fp,"%9f ",P[i]) ;
			
		        if((i+1)%Width==0)
				fprintf(fp,"\n");
		}
    		fclose(fp);
	}

	
	//printf("\n Matrix multiplication completed !!\n\n"); 
	free(M);
	free(N);
	free(P);

}


////////////////////////////////////////////////////////////////////////
//
//
////////////////////////////////////////////////////////////////////////


void fill(float* h, int w)
{ 
	
	unsigned int i, num;
	int divide;
	FILE *f;

	f=fopen("/dev/urandom", "r");
	if (!f) {
        	fprintf(stderr, "Failed open file\n");
        	exit(1);
    	}
	for(i=0; i< w*w; i++)
	{
		fread(&num, sizeof(unsigned int), 1, f);
		fread(&divide, sizeof(int), 1, f);
		h[i] = ((float)num)/((float)divide);
		//scanf("%f",&h[i]);
	}
	fclose(f);
/*
	unsigned int i;
	srand((unsigned int)time(NULL));
	for(i=0; i< w*w; i++)
	{
		h[i] = ((float)rand()/(float)(RAND_MAX)) * 99;
		//scanf("%f",&h[i]);
	}
	
*/

} 

////////////////////////////////////////////////////////////////////////
//
//
////////////////////////////////////////////////////////////////////////

void test_invert(int n )
{
    	
	//printf("Enter the order of the square matrix :");
	//scanf("%d",&n);
    	const int mybatch = 1;


	//float* mat1[n * n];
	float mat1_size = sizeof(float) * n * n;
    	float* mat1 = (float*) malloc(mat1_size);

	fill(mat1, n);

    	float *result_flat = (float *)malloc(mybatch*n*n*sizeof(float));
    	float **results = (float **)malloc(mybatch*sizeof(float *));

    	for (int i = 0; i < mybatch; i++)
      		results[i] = result_flat + (i*n*n);

    	float **inputs = (float **)malloc(mybatch*sizeof(float *));
    	
	//inputs[0]  = zero_pivot;

	inputs[0]  = mat1;


	invert(inputs, results, n, mybatch);

	if(n<11)
	{

		for (int qq = 0; qq < mybatch; qq++)
		{
	      		if(mybatch==1)
				fprintf(stdout, "Input Matrix, M :\n\n");
			else
				fprintf(stdout, "Input Matrix %d:\n\n", qq);
			
	      		for(int i=0; i<n; i++)
	      		{
	        		for(int j=0; j<n; j++)
				{	
					if(inputs[qq][i*n+j])
		            			fprintf(stdout,"%12f ",inputs[qq][i*n+j]);
					else
						fprintf(stdout,"%11f ",inputs[qq][i*n+j]);
				}
	        			fprintf(stdout,"\n");
	      		}
	    	}
	    	fprintf(stdout,"\n\n");




	    	for (int qq = 0; qq < mybatch; qq++)
		{

			if(mybatch==1)
				fprintf(stdout, "Inverse of the Input Matrix, Inv(M):\n\n");
			else
				fprintf(stdout, "Inverse Matrix %d:\n\n", qq);
	      		for(int i=0; i<n; i++)
	      		{
	        		for(int j=0; j<n; j++)
				{
					if(results[qq][i*n+j])
		            			fprintf(stdout,"%10f ",results[qq][i*n+j]);
					else
		            			fprintf(stdout,"%9f ",results[qq][i*n+j]);
	        		
				}
				fprintf(stdout,"\n");
	      		}
	    	}
	}


	else // order of the matrix is more than 10 x 10 then output the results in the file
	{
		printf("\nThe order of matrix is too large to display in terminal\n, Please open the file : Mat_Inv_out.txt located in the current folder. To see the output.\n\n");
		
		FILE *fp;


 		fp = fopen("Mat_Inv_out", "w");

 		if (!fp) 
		{
    			fprintf(stderr, "Failed to open Mat_Inv_out.\n");
		    	exit(1);
  		}



		for (int qq = 0; qq < mybatch; qq++)
		{

			if(mybatch==1)
				fprintf(fp, "Input Matrix , M:\n\n");
			else
				fprintf(fp, "Input Matrix %d:\n\n", qq);


	      	
			
	      		for(int i=0; i<n; i++)
      			{
        			for(int j=0; j<n; j++)
				{
					if(inputs[qq][i*n+j])
		            			fprintf(fp,"%12f ",inputs[qq][i*n+j]);
					else
						fprintf(fp,"%11f ",inputs[qq][i*n+j]);
				}
		            		
        			fprintf(fp,"\n");
	      		}
    		}
	    	fprintf(fp,"\n\n");

		for (int qq = 0; qq < mybatch; qq++)
		{
			if(mybatch==1)
				fprintf(fp, "Inverse of the Input Matrix, Inv(M):\n\n");
	      		else
				fprintf(fp, "Inverse %d:\n\n", qq);
	      		for(int i=0; i<n; i++)
	      		{
	        		for(int j=0; j<n; j++)	
				{
					if(results[qq][i*n+j])
		            			fprintf(fp,"%10f ",results[qq][i*n+j]);
					else
		            			fprintf(fp,"%9f ",results[qq][i*n+j]);
	        		
				}

	        		fprintf(fp,"\n");
	      		}
	    	}

		fclose(fp);
			
	}// end of if else condition for output

	float *A, *B;

	A=inputs[0];
	B=results[0];
	mul(A, B, n );

	//mul(inputs[0][], results[0][], n );

}

////////////////////////////////////////////////////////////////////////
//
//
////////////////////////////////////////////////////////////////////////

int main(int argc, char *argv[])
{
	if(argc!=2)
	{
		printf("Usage: %s <matrix_width>\n", argv[0]);
		return 0;
	}

	int w;
	w = atoi( argv[1] );
	
   	test_invert(w);
    	return 0;
}

/*
$ nvcc -arch=sm_20 -o t540 t540.cu -lcublas
$ ./t540
Input 0:

0.000000        3.000000        4.000000
1.000000        3.000000        10.000000
4.000000        9.000000        16.000000
Input 1:

0.500000        3.000000        4.000000
1.000000        3.000000        10.000000
4.000000        9.000000        16.000000
Input 2:

0.000000        3.000000        4.000000
1.000000        5.000000        6.000000
9.000000        8.000000        2.000000
Input 3:

22.000000       3.000000        4.000000
1.000000        5.000000        6.000000
9.000000        8.000000        2.000000


Inverse 0:

-0.700000       -0.200000       0.300000
0.400000        -0.266667       0.066667
-0.050000       0.200000        -0.050000
Inverse 1:

-1.076923       -0.307692       0.461538
0.615385        -0.205128       -0.025641
-0.076923       0.192308        -0.038462
Inverse 2:

-4.750000       3.250000        -0.250000
6.500000        -4.500000       0.500000
-4.625000       3.375000        -0.375000
Inverse 3:

0.045894        -0.031401       0.002415
-0.062802       -0.009662       0.154589
0.044686        0.179952        -0.129227
$


$ nvcc -arch=sm_20 -o t540 t540.cu -lcublas
$ ./t540 
Enter the order of the aquare matrix :4
Input 0:

-0.100222 -2.553872 -69.072723 0.016120 
-2.752346 -1.230871 1.997387 0.606710 
-0.029929 -0.583444 2.733107 0.254404 
-1.844285 -0.070541 1.906255 10.758991 


Inverse 0:

0.017501	-0.374555	0.713068	0.004234	
-0.056876	-0.005437	-1.457745	0.034861	
-0.012399	0.000729	0.052888	-0.001273	
0.004824	-0.064370	0.103305	0.094125	

*/
